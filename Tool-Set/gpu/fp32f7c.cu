

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 512
#define BLOCK_SIZE 256

__global__ void dot_product_float32(float *a, float *b, float *result) {
    __shared__ float temp[BLOCK_SIZE];
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    temp[tid] = (index < N) ? a[index] * b[index] : 0.0f;
    __syncthreads();

    // Reduce within block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            temp[tid] += temp[tid + stride];
        }
        __syncthreads();
    }

    // Store result of this block in global memory
    if (tid == 0) {
        atomicAdd(result, temp[0]);
    }
}

int main() {
    float *a, *b, *result;
    float *d_a, *d_b, *d_result;
    float final_result;

    // Allocate host memory
    a = (float*)malloc(N * sizeof(float));
    b = (float*)malloc(N * sizeof(float));
    result = (float*)malloc(sizeof(float));

    // Initialize host arrays
    for (int i = 0; i < N; ++i) {
        a[i] = 1.0f;
        b[i] = 1.0f;
    }
    *result = 0.0f;

    // Allocate device memory
    hipMalloc((void**)&d_a, N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));
    hipMalloc((void**)&d_result, sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dot_product_float32<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_a, d_b, d_result);

    // Copy result back to host
    hipMemcpy(result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    // Print result
    printf("Dot product (float32): %f\n", *result);

    // Free host memory
    free(a);
    free(b);
    free(result);

    return 0;
}

